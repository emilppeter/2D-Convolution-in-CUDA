#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <stdbool.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

int form_matrix(char input[],double mat[][500],int *i,int *j,bool test)
{
	int l=0,k=0;
	char temp[10000];
	int number_of_cols=0;
	while(input[k]!='\n')
    {  		
    	if(input[k]!=' ')
	    {
	   		temp[l]=input[k];
	   		l++;
	   		k++;
	   	}
    	else if (input[k]==' ') 
    	{
    		for (int z=l;z<10000;z++)
    			temp[z]=0;
    		mat[*i][*j]=atof(temp);
    		strcpy(temp," ");
    		l=0;
    		k++;
    		(*j)++;
    		number_of_cols++;    		
    	}
    }
   	if (input[k]=='\n' && test!=true)
   	{
   		for (int z=l;z<10000;z++)
    			temp[z]=0;
   		mat[*i][*j]=atof(temp);
    	strcpy(temp," ");
    	l=0;
    	k=0;
    	(*i)++;
    	*j=0;
    	number_of_cols++;
   	}
   	input[0]='\0';
   	return number_of_cols;
}
__global__ void
convolution(double *a, double *h, double *c,int c_rows, int c_cols,int z,int i,int j,int k)
{
  int n = blockIdx.x * blockDim.x + threadIdx.x; //idx
	int m = blockIdx.y * blockDim.y + threadIdx.y; //idy
  //printf("%d %d\n",m,n);
  if (m<c_rows && n<c_cols)
  {
    for(int p=0;p<=(j-1);p++)
   	{
   		for(int q=0;q<=(k-1);q++)
   		{
      if(!((m-p)<0 || (n-q)<0 || (m-p)>=z || (n-q)>=i))
      {
        c[(m*c_cols)+n]+=h[(p*k)+q]*a[((m-p)*i)+(n-q)];			
        __syncthreads();
      }
   		}
   	}
  }
}
int main(int argc, char **argv)
{
	FILE *read_file;
	char input[10000];
	int e=0,d=0,m=0,k=0,j=0,n=0,select=1,u=0,v=0;
	double a[500][500],h[500][500];
  hipError_t err = hipSuccess;
	int flag1=0,flag2=0;
  char *input_file;
  input_file=argv[1];
	read_file=fopen(input_file,"r");
	if (read_file==NULL)
	{
		printf("Error opening file\n");
		exit(1);
	}
	while(fgets(input,10000,read_file)) 
    {
    	bool test=false;

    	if (strcmp(input,"\n")==0)
    	{
    		select=2;
    		test=true;
    	}
    	if (select==1)
    	{
    		(m)++;
        if (test!=true && flag1==0)
        {
    		  n=form_matrix(input,a,&e,&d,test);
          flag1=1;
        }else
          form_matrix(input,a,&e,&d,test);
    	}
    	else if (select==2)
    	{
    		(j)++;
    		if (test!=true && flag2==0)
    		{
    			k=form_matrix(input,h,&u,&v,test);
    			flag2=1;
    		}
    		else 
    			form_matrix(input,h,&u,&v,test);
    	}
    	input[0]='\0';
    }
    --j;
    double *h_a=NULL,*h_h=NULL,*h_c=NULL;
   
    int c_rows=(m+j-1);
    int c_cols=(n+k-1);
    size_t size_a=(m*n)*sizeof(double);

    err=hipMallocManaged((void**)&h_a, size_a);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device matrix a (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    for (int i=0;i<m;i++)
    {
      for(int j=0;j<n;j++)
      {
        h_a[(i*n)+j]=a[i][j];
      }
    }

    size_t size_h=(j*k)*sizeof(double);

    err=hipMallocManaged((void**)&h_h, size_h);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device matrix h (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    for (int p=0;p<j;p++)
    {
      for(int q=0;q<k;q++)
      {
        h_h[(p*k)+q]=h[p][q];
      }
    }

    size_t size_c=(c_rows*c_cols)*sizeof(double);
    err=hipMallocManaged((void**)&h_c, size_c);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device matrix c (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    dim3 threadsPerBlock(32,32);
    dim3 numBlocks(c_rows/threadsPerBlock.x+1,c_cols/threadsPerBlock.y+1);
    //struct timeval begin, end;
    //gettimeofday(&begin, NULL);
    convolution<<<numBlocks, threadsPerBlock>>>(h_a,h_h, h_c,c_rows, c_cols, m, n, j, k);
    //gettimeofday(&end, NULL);
    err = hipGetLastError();

    //int time_in_us = 1e6*(end.tv_sec-begin.tv_sec) + (end.tv_usec-begin.tv_usec);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
   err = hipDeviceSynchronize();
  
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to synchronize the device (error code: %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
   	
   	for (int i=0;i<(m+j-1);i++)
    {
      for(int z=0;z<(n+k-1);z++)
      {
        printf("%0.3lf ",h_c[(i*c_cols)+z]);
      }
      printf("\n");
    }
    //printf("Time for V2 Kernel = %d us\n", time_in_us);
    err = hipFree(h_a);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device matrix A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
  
    err = hipFree(h_h);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device matrix h (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(h_c);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device matrix c (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    fclose(read_file);
	return 0;
}